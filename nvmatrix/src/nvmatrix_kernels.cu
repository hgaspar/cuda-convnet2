#include "hip/hip_runtime.h"
/*
 * Copyright 2014 Google Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <hip_runtime.h>
#include "../include/nvmatrix_kernels.cuh"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    const int numThreads = hipBlockDim_x * hipGridDim_x;
    //    const unsigned int numEls = tgtWidth * tgtHeight;
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}

__global__ void kDotProduct_r(float* a, float* b, float* target,  const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * hipBlockIdx_x + hipThreadIdx_x;
    shmem[hipThreadIdx_x] = 0;
    if (eidx < hipGridDim_x * DP_BLOCKSIZE) {
        for (; eidx < numElements; eidx += hipGridDim_x * DP_BLOCKSIZE) {
            shmem[hipThreadIdx_x] += a[eidx] * b[eidx];
        }
    }
    __syncthreads();
    if (hipThreadIdx_x < 256) {
        shmem[hipThreadIdx_x] += shmem[hipThreadIdx_x + 256];
    }
    __syncthreads();
    if (hipThreadIdx_x < 128) {
        shmem[hipThreadIdx_x] += shmem[hipThreadIdx_x + 128];
    }
    __syncthreads();
    if (hipThreadIdx_x < 64) {
        shmem[hipThreadIdx_x] += shmem[hipThreadIdx_x + 64];
    }
    __syncthreads();
    if (hipThreadIdx_x < 32) {
        volatile float* mysh = &shmem[hipThreadIdx_x];
        *mysh += mysh[32];
        *mysh += mysh[16];
        *mysh += mysh[8];
        *mysh += mysh[4];
        *mysh += mysh[2];
        *mysh += mysh[1];
        if (hipThreadIdx_x == 0) {
            target[hipBlockIdx_x] = *mysh;
        }
    }
}

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * hipBlockIdx_x + hipThreadIdx_x;
    /* Each thread gets same seed, a different sequence number,
     no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}

